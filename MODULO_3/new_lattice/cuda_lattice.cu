#include "hip/hip_runtime.h"
#include "cuda_lattice.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>

__host__ void cudaInitFromLattice() {
	grid = dim3(iDivUp(length, 2 * BLOCK_SIDE));
	
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_length), &length, sizeof(int)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_beta), &beta, sizeof(double)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_p_length), &p_length, sizeof(int)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_nparticles), &nparticles, sizeof(int)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_y), y, sizeof(y)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_links), links, sizeof(links)));
	
	/* gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_rules), rules, sizeof(rules)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_repetitions), repetitions, sizeof(repetitions)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_nrules), &nrules, sizeof(int))); */
	
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_obs1), obs1, sizeof(obs1)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_obs2), obs2, sizeof(obs2)));
	
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_pars), pars, sizeof(pars)));
    
	double* temp_delta_obs1;
	gpuErrchk(hipGetSymbolAddress((void **)&temp_delta_obs1, d_delta_obs1));
	gpuErrchk(hipMemset(temp_delta_obs1, 0, sizeof(y)));
	double* temp_delta_obs2;
	gpuErrchk(hipGetSymbolAddress((void **)&temp_delta_obs2, d_delta_obs2));
	gpuErrchk(hipMemset(temp_delta_obs2, 0, sizeof(y)));
	
	func_ds = harm_pot;
}

__host__ void cudaRetrieveLattice() {
	gpuErrchk(hipMemcpyFromSymbol(y, HIP_SYMBOL(d_y), sizeof(y)));
}

__host__ void cudaDestroyLattice() {
	hipDeviceReset();
}

// 2) delta + GPU
__global__ void metropolisStepGPU(int bw, int dead_site, int length_, Function_ds func) {	
	// thread position
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	// spin position
	int jj = 2 * j + (bw % 2);
	if(jj >= length_) return;
	
	if(cuda_index1d(jj,length_) == dead_spin) return; // avoid loops!
	
	double eta = d_beta / d_p_length;
	double delta = 2. * sqrt(eta);
	
	double y0 = d_y[cuda_index1d(jj,length_)]; //centre
	double y1 = d_y[d_links[2*jj+0]]; //left
	double y2 = d_y[d_links[2*jj+1]]; //right
	double yp = (d_rr[cuda_index1d(jj,length_) * 2. - 1.) * delta + y0;
	
	double outputs[2];
	double ds = func(y0, y1, y2, yp, d_pars, outputs);
	double r = exp(-ds);
	if (d_rr[cuda_index1d(jj,length_)+length_] < r) {
        d_y[cuda_index1d(jj,length_)] = yp;
		d_delta_obs1[cuda_index1d(jj,length_)] += outputs[0];
		d_delta_obs2[cuda_index1d(jj,length_)] += outputs[1];
    }
}

__host__ void cudaUpdateMetropolis() {
	double rr[2*length];
	for(int ii = 0; ii < 2*length; ii++) rr[ii] = gen.randF();
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_rr), rr, sizeof(rr)));
	int dead_site = gen.randL(0, length);
    
	metropolisStepGPU<<<grid, thread_block>>>(0, dead_site, length, func_ds);
	gpuErrchk(hipDeviceSynchronize());
	metropolisStepGPU<<<grid, thread_block>>>(1, dead_site, length, func_ds);
	gpuErrchk(hipDeviceSynchronize());
}

__host__ void calculateObsGPU() {	
	double h_delta_obs1[length], h_delta_obs2[length];
	gpuErrchk(hipMemcpyFromSymbol(h_delta_obs1, HIP_SYMBOL(d_delta_obs1), length * sizeof(double)));
	gpuErrchk(hipMemcpyFromSymbol(h_delta_obs2, HIP_SYMBOL(d_delta_obs2), length * sizeof(double)));
	
	double normalization = 1. / p_length;
	for(int i = 0; i < nparticles; i++) {
		double sum1 = 0., sum2 = 0.;
		for(int j = 0; j < p_length; j++) {
			sum1 += h_delta_obs1[j + p_length * i];
			sum2 += h_delta_obs2[j + p_length * i];
		}
		obs1[i] += sum1 * normalization;
		obs2[i] += sum2 * normalization;
	}
	
	double* temp_delta_energy;
    gpuErrchk(hipGetSymbolAddress((void **)&temp_delta_energy, d_delta_energy));
    gpuErrchk(hipMemset(temp_delta_energy, 0, sizeof(y)));
    double* temp_delta_magnetization;
    gpuErrchk(hipGetSymbolAddress((void **)&temp_delta_magnetization, d_delta_magnetization));
    gpuErrchk(hipMemset(temp_delta_magnetization, 0, sizeof(y)));
}