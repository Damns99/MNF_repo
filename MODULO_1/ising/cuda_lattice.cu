#include "hip/hip_runtime.h"
#include "lattice.h"

#include <hip/hip_runtime.h>

__global__ void squareMetropolisStepGPU(Lattice2D* lattice, double* rr, int size, int bw, int dead_spin) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index >= size) return;
	int x = 2 * index + ((2 * index / lattice->length + bw) % 2);
	if(x == dead_spin) return; // avoid loops!
	int newspin = - lattice->spin[x];
	double denergy = -2. * newspin * lattice->extrafield;
	for (int j = 0; j < lattice->links_per_spin; j++) {
		denergy += -2. * JACC * newspin * lattice->spin[lattice->links[lattice->links_per_spin * x + j]];
	}
	double r = exp(-lattice->beta * denergy);
	if (rr[index] < r) lattice->spin[x] = newspin;
}

void cudaSquareUpdateMetropolis(Lattice2D* lattice) { // accettanza?
	assert(lattice->length % 2 == 0);
	int nbw = lattice->length * lattice->length / 2;
	int sizedbl = nbw * sizeof(double);
	double rrb[nbw], rrw[nbw];
	for(int ii = 0; ii < nbw; ii++) {
		rrb[ii] = lattice->gen.randF();
		rrw[ii] = lattice->gen.randF();
	}
	int dead_spin = lattice->gen.randL(0, lattice->length * lattice->length);
	
	double* d_rrb;
	double* d_rrw;
	Lattice2D* d_lattice;
	hipMalloc((void **)&d_rrb, sizedbl);
	hipMalloc((void **)&d_rrw, sizedbl);
	hipMalloc((void **)&d_lattice, sizeof(Lattice2D));
	
	hipMemcpy(d_rrb, rrb, sizedbl, hipMemcpyHostToDevice);
	hipMemcpy(d_rrw, rrw, sizedbl, hipMemcpyHostToDevice);
	hipMemcpy(d_lattice, lattice, sizeof(Lattice2D), hipMemcpyHostToDevice);
	
	int nthreads = nbw > 256 ? 256 : nbw;
	int nblocks = (nbw - 1) / nthreads + 1;
	squareMetropolisStepGPU<<<nblocks,nthreads>>>(d_lattice, d_rrb, nbw, 0, dead_spin);
	hipDeviceSynchronize();
	squareMetropolisStepGPU<<<nblocks,nthreads>>>(d_lattice, d_rrw, nbw, 1, dead_spin);
	hipDeviceSynchronize();
	
	hipMemcpy(lattice, d_lattice, sizeof(Lattice2D), hipMemcpyDeviceToHost);
	lattice->calculateEnergyMagnetizaton();
	
	hipFree(d_rrb);
	hipFree(d_rrw);
	hipFree(d_lattice);
}