#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, std::string file, int line, bool abort=false)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file.c_str(), line);
      if (abort) exit(code);
   }
}

class myClass {
	public:
		int* d_value;
		size_t freem = 1, totm = 1;
		
		__host__ myClass() {;}
		__host__ ~myClass() {;}
		__host__ void initClass(int* init);
		__host__ void destroyClass();
		__host__ void runClass();
		__host__ void retrieveClass(int* h_value);
};

__global__ void setOne(myClass c) {
	*c.d_value = 1;
}

__host__ void myClass::initClass(int* init) {
	gpuErrchk(hipMemGetInfo(&freem, &totm));
	std::cout << "free: " << freem << " total: " << totm << std::endl;
	gpuErrchk(hipMalloc((void **)&d_value, sizeof(int)));
	gpuErrchk(hipMemcpy(d_value, init, sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemGetInfo(&freem, &totm));
	std::cout << "free: " << freem << " total: " << totm << std::endl;
}
__host__ void myClass::destroyClass() {
	gpuErrchk(hipMemGetInfo(&freem, &totm));
	std::cout << "free: " << freem << " total: " << totm << std::endl;
	gpuErrchk(hipFree(d_value));
	gpuErrchk(hipMemGetInfo(&freem, &totm));
	std::cout << "free: " << freem << " total: " << totm << std::endl;
}
__host__ void myClass::runClass() {
	setOne<<<1,1>>>(*this);
	gpuErrchk(hipDeviceSynchronize());
}
__host__ void myClass::retrieveClass(int* h_value) {
	gpuErrchk(hipMemGetInfo(&freem, &totm));
	std::cout << "free: " << freem << " total: " << totm << std::endl;
	gpuErrchk(hipMemcpy(h_value, d_value, sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemGetInfo(&freem, &totm));
	std::cout << "free: " << freem << " total: " << totm << std::endl;
}

int main() {
	int a = 101;
	myClass c;
	c.initClass(&a);
	c.runClass();
	c.retrieveClass(&a);
	c.destroyClass();
}